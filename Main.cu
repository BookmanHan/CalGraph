#include "hip/hip_runtime.h"
#include "Import.hpp"
#include "CalGraph.hpp"
#include "MathFunction.hpp"
#include "Operation.hpp"
#include "Objective.hpp"

using namespace cal;
using namespace cal::math;
using namespace cal::objective;

int main(int argc, char **argv)
{
	af::array input = af::randn(10, 10);
	input(af::seq(0, af::end, 2), af::span) += 10.0f;

	af::array output = af::randn(10, 2);
	output(af::seq(0, af::end, 2), af::span) = 0.9;
	output(af::seq(1, af::end, 2), af::span) = 0.1;

	af::array hids = af::constant(0.f, 10, 10);
	af::array bitmask = af::constant(0, 10, 10, 30, u8);
	bitmask(af::span, af::span, af::seq(0, 20)) = 1;
	
	CalGraph cg;

	autoref x = cg.datum(input);
	autoref y = cg.datum(output);
	autoref mask = cg.datum(bitmask);

	autoref W1 = cg.variable_xavier(10, 10);
	autoref W2 = cg.variable_xavier(10, 10);
	autoref W3 = cg.variable_xavier(10, 2);

	auto hidden = &(cg.datum(hids));
	auto loss = &(cg.datum(af::constant(0.f, 10, 2)));

	int n = 0;
	for(int i=0; i<20; ++i)
	{
		//autoref step = cg.datum(n++);
		hidden = &(tanh(x * W1 + (*hidden) * W2));
		//hidden = &(*hidden % slice(3, mask, step));

		autoref rep = (*hidden) * W3;

		loss = &(*loss + (rep - y) % (rep - y));
	}

	cg.loss(*loss, "RNN");
	cg.train(10000);

	return 0;
}