#include "hip/hip_runtime.h"
#include "Import.hpp"
#include "CalGraph.hpp"
#include "MathFunction.hpp"
#include "LogicFunction.hpp"
#include "Operation.hpp"
#include "Objective.hpp"

//TODO:
//1.�Ѽ���ͼ����ȱ�����Ϊ��ȱ�����
//2.�� ArrayFire ÿһ����������Ӧ������ͼ�ϡ�

using namespace cal;
using namespace cal::math;
using namespace cal::objective;
using namespace cal::logic;

int main(int argc, char **argv)
{
	int n_sample = 1000;
	int n_feature = 300;
	int n_word = 100;
	int n_step = 5;
	int n_output = n_word;

	af::array input = af::randu(n_sample, n_step) * (n_word - 1);
	input = af::ceil(input);

	af::array output = af::randu(n_sample, n_step) * (n_word - 1);
	input = af::ceil(input);

	af::array hids = af::constant(0.f, n_sample, n_feature);
	af::array bitmask = af::constant(0, n_sample, n_feature, n_step);
	bitmask(af::span, af::span, af::seq(0, n_step/2)) = 1;

	CalGraph cg;

	autoref x = cg.datum(input);
	autoref y = cg.datum(output);
	autoref mask = cg.datum(bitmask);

	autoref Em = cg.variable_embedding(af::randn(n_word, n_feature));
	autoref W1 = cg.variable_xavier(n_feature, n_feature);
	autoref W2 = cg.variable_xavier(n_feature, n_feature);
	autoref W4 = cg.variable_xavier(n_feature, n_feature);
	autoref W5 = cg.variable_xavier(n_feature, n_feature);
	autoref W6 = cg.variable_xavier(n_feature, n_word);
	autoref W7 = cg.variable_xavier(n_feature, n_feature);
	autoref W8 = cg.variable_xavier(n_feature, n_step);

	auto hidden = &(cg.datum(hids));
	auto loss = &(cg.datum(af::constant(0.f, n_sample)));

	vector<Symbol*>	hidden_units;
	int n = 0;
	for(int i=0; i < n_step; ++i)
	{
		autoref step = cg.datum(n++);
		hidden = &(tanh(embed(Em, slice(2, x, step), false) * W1 + (*hidden) * W2));
		hidden = &(*hidden % slice(3, mask, step));
		hidden_units.push_back(hidden);
	}

	autoref hidden_step = join_step(hidden_units);

	auto decoder_word = &(cg.datum(af::constant(0, n_sample, s32)));
	int m = 0;
	for (int i = 0; i < n_step; ++i)
	{
		autoref step = cg.datum(m++);
		autoref attention = softmax(tanh(*hidden * W8));
		autoref context = weight_step(attention, hidden_step);

		hidden = &(tanh(context * W7 + embed(Em, *decoder_word) * W4));
		autoref prob = dim(softmax(*hidden * W6));
		decoder_word = &(max_index(*hidden));
		cg.loss(
			scalar_sum(cross_entropi(hoc(n_sample, n_word, slice(2, x, step)), prob)), "RNN");
	}

	cg.train(10000,
		[&](int epos)
	{
		af::array input = af::randu(n_sample, n_step) * (n_word - 1);
		input = af::ceil(input);

		af::array output = af::randu(n_sample, n_step) * (n_word - 1);
		input = af::ceil(input);

		x.set(input);
		y.set(output);
	});

	return 0;
}