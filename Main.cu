#include "hip/hip_runtime.h"
#include "Import.hpp"
#include "CalGraph.hpp"
#include "MathFunction.hpp"
#include "Operation.hpp"
#include "Objective.hpp"

using namespace cal;
using namespace cal::math;
using namespace cal::objective;

int main(int argc, char **argv)
{
	int n_sample = 1000;
	int n_feature = 100;

	af::array input = af::randn(n_sample, n_feature);
	input(af::seq(0, af::end, 2), af::span) += 10.0f;

	af::array output = af::randn(n_sample, 2);
	output(af::seq(0, af::end, 2), af::span) = 0.9;
	output(af::seq(1, af::end, 2), af::span) = 0.1;

	af::array hids = af::constant(0.f, n_sample, n_feature);
	
	CalGraph cg;

	autoref x = cg.datum(input);
	autoref y = cg.datum(output);

	autoref W1 = cg.variable_xavier(n_feature, n_feature);
	autoref W2 = cg.variable_xavier(n_feature, n_feature);
	autoref W3 = cg.variable_xavier(n_feature, 2);

	auto hidden = &(cg.datum(hids));
	auto loss = &(cg.datum(af::constant(0.f, n_sample, 2)));

	int n = 0;
	for(int i=0; i<15; ++i)
	{
		hidden = &(tanh(x * W1 + (*hidden) * W2));
		autoref rep = softmax((*hidden) * W3);
		loss = &(*loss + cross_entropi(rep, y));
	}

	cg.loss(*loss, "RNN");
	cg.train(10000);

	return 0;
}