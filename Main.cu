#include "hip/hip_runtime.h"
#include "Import.hpp"
#include "CalGraph.hpp"
#include "MathFunction.hpp"
#include "LogicFunction.hpp"
#include "Operation.hpp"
#include "Objective.hpp"

using namespace cal;
using namespace cal::math;
using namespace cal::objective;
using namespace cal::logic;

int main(int argc, char **argv)
{
	int n_sample = 3000;
	int n_feature = 100;
	int n_word = 100;
	int n_step = 20;
	int n_output = n_word;

	af::array input = af::randu(n_sample, n_step) * (n_word - 1);
	input = af::ceil(input);

	af::array output = af::randu(n_sample, n_step) * (n_word - 1);
	input = af::ceil(input);

	af::array hids = af::constant(0.f, n_sample, n_feature);
	af::array bitmask = af::constant(0, n_sample, n_feature, n_step);
	bitmask(af::span, af::span, af::seq(0, n_step/2)) = 1;

	CalGraph cg;

	autoref x = cg.datum(input);
	autoref y = cg.datum(output);
	autoref mask = cg.datum(bitmask);

	autoref Em = cg.variable_embedding(af::randn(n_word, n_feature));
	autoref W1 = cg.variable_xavier(n_feature, n_feature);
	autoref W2 = cg.variable_xavier(n_feature, n_feature);
	autoref W4 = cg.variable_xavier(n_feature, n_feature);
	autoref W5 = cg.variable_xavier(n_feature, n_feature);
	autoref W6 = cg.variable_xavier(n_feature, n_word);
	autoref W7 = cg.variable_xavier(n_feature, n_feature);

	auto hidden = &(cg.datum(hids));
	auto loss = &(cg.datum(af::constant(0.f, n_sample)));

	int n = 0;
	for(int i=0; i < n_step; ++i)
	{
		autoref step = cg.datum(n++);
		hidden = &(tanh(embed(Em, slice(2, x, step)) * W1 + (*hidden) * W2));
		hidden = &(*hidden % slice(3, mask, step));;
	}

	auto decoder_word = &(cg.datum(af::constant(0, n_sample, s32)));
	int m = 0;
	for (int i = 0; i < n_step; ++i)
	{
		autoref step = cg.datum(m++);
		hidden = &(tanh(*hidden * W7 + embed(Em, *decoder_word) * W4));
		autoref prob = softmax(*hidden * W6);
		decoder_word = &(max_index(*hidden));
		loss = &(*loss + 
			scalar_sum(cross_entropi(hoc(n_sample, n_word, slice(2, x, step)), prob)));
	}

	cg.loss(*loss, "RNN");

	cg.train(10000,
		[&](int epos)
	{
		x.set(input);
		y.set(output);
	});


	return 0;
}