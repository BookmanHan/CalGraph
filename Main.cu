#include "hip/hip_runtime.h"
#include "Import.hpp"
#include "CalGraph.hpp"
#include "MathFunction.hpp"
#include "LogicFunction.hpp"
#include "Operation.hpp"
#include "Objective.hpp"

using namespace cal;
using namespace cal::math;
using namespace cal::objective;
using namespace cal::logic;

int main(int argc, char **argv)
{
	int n_sample = 1000;
	int n_feature = 100;
	int n_word = 100;
	int n_step = 20;
	int n_output = 2;

	af::array input = af::randu(n_sample, n_step) * (n_word - 1);
	input = af::ceil(input);

	af::array output = af::randu(n_sample, n_output);
	output(output > 0.5) = 0.95;
	output(output < 0.5) = 0.05;

	af::array hids = af::constant(0.f, n_sample, n_feature);
	af::array bitmask = af::constant(0, n_sample, n_feature, n_step, u8);
	bitmask(af::span, af::span, af::seq(0, 15)) = 1;

	CalGraph cg;

	autoref x = cg.datum(input);
	autoref y = cg.datum(output);
	autoref mask = cg.datum(bitmask);

	autoref Em = cg.variable_embedding(af::randn(n_word, n_feature));
	autoref W1 = cg.variable_xavier(n_feature, n_feature);
	autoref W2 = cg.variable_xavier(n_feature, n_feature);
	autoref W3 = cg.variable_xavier(n_word, 2);
	autoref W4 = cg.variable_xavier(n_feature, n_feature);
	autoref W5 = cg.variable_xavier(n_feature, n_feature);
	autoref W6 = cg.variable_xavier(n_feature, n_word);
	autoref W7 = cg.variable_xavier(n_feature, n_feature);

	auto hidden = &(cg.datum(hids));
	auto loss = &(cg.datum(af::constant(0.f, n_sample, 2)));

	int n = 0;
	for(int i=0; i < 5; ++i)
	{
		autoref step = cg.datum(n++);
		hidden = &(tanh(embed(Em, slice(2, x, step)) * W1 + (*hidden) * W2));
		hidden = &(*hidden % slice(3, mask, step));;
	}

	auto decoder_word = &(cg.datum(af::constant(0, n_sample, s32)));
	int m = 0;
	for (int i = 0; i < 5; ++i)
	{
		autoref step = cg.datum(m++);
		hidden = &(*hidden * W7 + embed(Em, *decoder_word) * W4);
		autoref prob = *hidden * W6;
		decoder_word = &(max_index(prob));
		print(*decoder_word);

		loss = &(*loss + cross_entropi(prob*W3, y));
	}

	cg.loss(*loss, "RNN");

	try
	{
		cg.train(10000,
			[&](int epos)
		{
			x.set(input);
			y.set(output);
		});
	}
	catch (af::exception e)
	{
		cout << e.what() << endl;
	}

	return 0;
}