#include "hip/hip_runtime.h"
#include "Import.hpp"
#include "CalGraph.hpp"
#include "MathFunction.hpp"
#include "Operation.hpp"
#include "Objective.hpp"

using namespace cal;
using namespace cal::math;
using namespace cal::objective;

int main(int argc, char **argv)
{
	int n_sample = 10000;
	int n_feature = 100;
	int n_step = 20;

	af::array input = af::randu(n_sample, n_step) * 10.0f;
	input = af::ceil(input);

	af::array output = af::randn(n_sample, 2);
	output(af::seq(0, af::end, 2), af::span) = 0.9;
	output(af::seq(1, af::end, 2), af::span) = 0.1;

	af::array hids = af::constant(0.f, n_sample, n_feature);
	af::array bitmask = af::constant(0, n_sample, n_feature, 20, u8);
	bitmask(af::span, af::span, af::seq(0, 15)) = 1;

	CalGraph cg;

	autoref x = cg.datum(input);
	autoref y = cg.datum(output);
	autoref mask = cg.datum(bitmask);

	autoref Em = cg.variable_embedding(af::randn(20, 100));
	autoref W1 = cg.variable_xavier(n_feature, n_feature);
	autoref W2 = cg.variable_xavier(n_feature, n_feature);
	autoref W3 = cg.variable_xavier(n_feature, 2);
	autoref W4 = cg.variable_xavier(n_feature, n_feature);
	autoref W5 = cg.variable_xavier(n_feature, n_feature);

	auto hidden = &(cg.datum(hids));
	auto loss = &(cg.datum(af::constant(0.f, n_sample, 2)));

	int n = 0;
	for(int i=0; i<20; ++i)
	{
		autoref step = cg.datum(n++);
		hidden = &(tanh(embed(Em, slice(2, x, step)) * W1 + (*hidden) * W2));
		hidden = &(*hidden % slice(3, mask, step));;
	}

	int m = 0;
	for (int i = 0; i < 20; ++i)
	{
		autoref step = cg.datum(m++);
		autoref rep = softmax(((*hidden) * W5 + embed(Em, slice(2, x, step)) * W4) * W3);
		loss = &(*loss + cross_entropi(rep, y));
	}

	cg.loss(*loss, "RNN");

	try
	{
		cg.train(10000,
			[&](int epos)
		{
			x.set(input);
			y.set(output);
		});
	}
	catch (af::exception e)
	{
		cout << e.what() << endl;
	}

	return 0;
}